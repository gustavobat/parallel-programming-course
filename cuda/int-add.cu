#include "hip/hip_runtime.h"
#include "common.h"
#include <iostream>

__global__ void intAddKernel(const int *a, const int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int h_A = 4;
    int h_B = 123;
    int h_C;

    int *d_A, *d_B, *d_C;

    CHECK_ERROR(hipMalloc((void **)&d_A, sizeof(int)));
    CHECK_ERROR(hipMalloc((void **)&d_B, sizeof(int)));
    CHECK_ERROR(hipMalloc((void **)&d_C, sizeof(int)));

    CHECK_ERROR(hipMemcpy(d_A, &h_A, sizeof(int), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_B, &h_B, sizeof(int), hipMemcpyHostToDevice));
    intAddKernel<<<1, 1>>>(d_A, d_B, d_C);
    CHECK_ERROR(hipMemcpy(&h_C, d_C, sizeof(int), hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(d_A));
    CHECK_ERROR(hipFree(d_B));
    CHECK_ERROR(hipFree(d_C));

    printf("%d + %d = %d!\n", h_A, h_B, h_C);
    return 0;
}
