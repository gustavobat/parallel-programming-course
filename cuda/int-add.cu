
#include <hip/hip_runtime.h>
#include <iostream>

// Macro for checking cuda errors following a cuda launch or api call
static void checkCudaErrors(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
        exit(EXIT_FAILURE);
    }
}
#define CHECK_ERROR(err) (checkCudaErrors(err, __FILE__, __LINE__))

__global__ void intAddKernel(const int *a, const int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int h_A = 2;
    int h_B = 2;
    int h_C;

    int *d_A, *d_B, *d_C;

    CHECK_ERROR(hipMalloc((void**)&d_A, sizeof(int)));
    CHECK_ERROR(hipMalloc((void**)&d_B, sizeof(int)));
    CHECK_ERROR(hipMalloc((void**)&d_C, sizeof(int)));

    CHECK_ERROR(hipMemcpy(d_A, &h_A, sizeof(int), hipMemcpyHostToDevice));
    CHECK_ERROR(hipMemcpy(d_B, &h_B, sizeof(int), hipMemcpyHostToDevice));
    intAddKernel<<<1, 1>>>(d_A, d_B, d_C);
    CHECK_ERROR(hipMemcpy(&h_C, d_C, sizeof(int), hipMemcpyDeviceToHost));

    CHECK_ERROR(hipFree(d_A));
    CHECK_ERROR(hipFree(d_B));
    CHECK_ERROR(hipFree(d_C));

    printf("%d + %d = %d!\n", h_A, h_B, h_C);
    return 0;
}
